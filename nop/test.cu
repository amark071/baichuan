   #include <hip/hip_runtime.h>
                                                                                                                                                              
   #include <stdio.h>
   extern "C" void cu(int* output, int input);
   
int main() {
    int input = 5;
    int* output;
    int* d_output;
        hipMalloc(&d_output, sizeof(int) * 16);
        output = (int*)malloc(sizeof(int) * 16);
        void* args[] = { &d_output, &input };
        hipFunction_t function;
        hipModule_t module;
        hipModuleLoad(&module, "cu.cubin");
        hipModuleGetFunction(&function, module, "_Z12simpleKernelPii");
        hipModuleLaunchKernel(function, 32, 1, 1, 32, 1, 1, 16384, 0, args, 0);
        hipMemcpy(output, d_output, sizeof(int) * 16, hipMemcpyDeviceToHost);
        for (int i = 0; i < 16; i++) {
          printf("output[%d] = %d\n", i, output[i]);
        }
        free(output);
        hipFree(d_output);
        hipModuleUnload(module);
    return 0;
  }