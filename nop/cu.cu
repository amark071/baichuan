#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void simpleKernel(int* output, int input) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    output[tid] = input * 2;
}
int main() {
    int input = 5;
    int* output;
    int* d_output;
    hipMalloc(&d_output, sizeof(int) * 16);
    output = (int*)malloc(sizeof(int) * 16);
    simpleKernel<<<4, 4>>>(d_output, input);
    hipMemcpy(output, d_output, sizeof(int) * 16, hipMemcpyDeviceToHost);
    for (int i = 0; i < 16; i++) {
        printf("output[%d] = %d\n", i, output[i]);
    }
    free(output);
    hipFree(d_output);
    return 0;
}