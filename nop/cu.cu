#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void simpleKernel(int* output, int input) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    output[tid] = input * 2;
}

int main() {
    int input = 5;
    int* output;
    int* d_output;

    // 在设备上分配内存
    hipMalloc(&d_output, sizeof(int) * 1024);

    // 在主机上分配内存
    output = (int*)malloc(sizeof(int) * 1024);

    // 在设备上启动内核函数
    simpleKernel<<<32, 32>>>(d_output, input);

    // 将结果从设备拷贝到主机
    hipMemcpy(output, d_output, sizeof(int) * 1024, hipMemcpyDeviceToHost);

    // 打印结果
    for (int i = 0; i < 10; i++) {
        printf("output[%d] = %d\n", i, output[i]);
    }

    // 释放内存
    free(output);
    hipFree(d_output);

    return 0;
}