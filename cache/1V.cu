#include <iostream>
#include <hip/hip_runtime.h>


// 检查CUDA错误的宏
#define checkCudaErrors(err) { \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error: " << err << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

int main() {
    hipDevice_t device;
    hipCtx_t context;
    hipModule_t module;
    hipFunction_t kernel;
    hipError_t err;

    // 初始化CUDA驱动API
    err = hipInit(0);
    if(err==hipSuccess) std::cout<<1<<' ';
    checkCudaErrors(err);

    // 获取第一个设备
    err = hipDeviceGet(&device, 0);
    if(err==hipSuccess) std::cout<<2<<' ';
    checkCudaErrors(err);

    // 创建上下文
    err = hipCtxCreate(&context, 0, device);
    if(err==hipSuccess) std::cout<<3<<' ';
    checkCudaErrors(err);

    // 加载模块
    err = hipModuleLoad(&module, "matmul_cache_kernel.cubin");
    if(err==hipSuccess) std::cout<<4<<' ';
    checkCudaErrors(err);

    // 获取内核函数
    err = hipModuleGetFunction(&kernel, module, "matmul_cache_kernel_0d1d2d3d4d5d6d7c8d9c10d11c");
    if(err==hipSuccess) std::cout<<5<<' ';
    checkCudaErrors(err);

    // 定义输入和输出数据
    float h_A[32];
    float h_B[32];
    float h_C[32];
    float *d_A, *d_B, *d_C;

    int M=32;
    int N=32;
    int K=32;
    int *d_m,*d_n,*d_k;

    int stride_am=32;
    int stride_ak=1;
    int stride_bk=32;
    int stride_bn=1;
    int stride_cm=32;
    int stride_cn=1;

    int *d_stride_am,*d_stride_ak,*d_stride_bk,*d_stride_bn,*d_stride_cm,*d_stride_cn;

    // 初始化输入数据
    for (int i = 0; i < M; i++) 
    {
        
            h_A[i] = static_cast<float>(i);       
    }
    for (int i = 0; i < K; i++) 
    {
            h_B[i]= static_cast<float>(i);       
    }
    
    // 分配设备内存
    err = hipMalloc((hipDeviceptr_t*)&d_A, M * sizeof(float));
    if(err==hipSuccess) std::cout<<6<<' ';
    checkCudaErrors(err);
    err = hipMalloc((hipDeviceptr_t*)&d_B, M * sizeof(float));
    checkCudaErrors(err);
    err = hipMalloc((hipDeviceptr_t*)&d_C, M * sizeof(float));
    checkCudaErrors(err);

    err = hipMalloc((hipDeviceptr_t*)&d_m, sizeof(int));
    if(err==hipSuccess) std::cout<<7<<' ';
    checkCudaErrors(err);
    err = hipMalloc((hipDeviceptr_t*)&d_n, sizeof(int));
    checkCudaErrors(err);
    err = hipMalloc((hipDeviceptr_t*)&d_k, sizeof(int));
    checkCudaErrors(err);

    err = hipMalloc((hipDeviceptr_t*)&d_stride_am, sizeof(int));
    if(err==hipSuccess) std::cout<<8<<' ';
    checkCudaErrors(err);
    err = hipMalloc((hipDeviceptr_t*)&d_stride_ak, sizeof(int));
    checkCudaErrors(err);
    err = hipMalloc((hipDeviceptr_t*)&d_stride_bk, sizeof(int));
    checkCudaErrors(err);
    err = hipMalloc((hipDeviceptr_t*)&d_stride_bn, sizeof(int));
    checkCudaErrors(err);
    err = hipMalloc((hipDeviceptr_t*)&d_stride_cm, sizeof(int));
    checkCudaErrors(err);
    err = hipMalloc((hipDeviceptr_t*)&d_stride_cn, sizeof(int));
    checkCudaErrors(err);


    // 将数据从主机复制到设备
    err = hipMemcpyHtoD((hipDeviceptr_t)d_A, h_A, M * sizeof(float));
    if(err==hipSuccess) std::cout<<9<<' ';
    checkCudaErrors(err);
    err = hipMemcpyHtoD((hipDeviceptr_t)d_B, h_B, M * sizeof(float));
    checkCudaErrors(err);

    err = hipMemcpyHtoD((hipDeviceptr_t)d_m, &M, sizeof(int));
    if(err==hipSuccess) std::cout<<10<<' ';
    checkCudaErrors(err);
    err = hipMemcpyHtoD((hipDeviceptr_t)d_n, &N, sizeof(int));
    checkCudaErrors(err);
    err = hipMemcpyHtoD((hipDeviceptr_t)d_k, &K, sizeof(int));
    checkCudaErrors(err);

    err = hipMemcpyHtoD((hipDeviceptr_t)d_stride_am, &stride_am, sizeof(int));
    if(err==hipSuccess) std::cout<<11<<' ';
    checkCudaErrors(err);
    err = hipMemcpyHtoD((hipDeviceptr_t)d_stride_ak, &stride_ak, sizeof(int));
    checkCudaErrors(err);
    err = hipMemcpyHtoD((hipDeviceptr_t)d_stride_bk, &stride_bk, sizeof(int));
    checkCudaErrors(err);
    err = hipMemcpyHtoD((hipDeviceptr_t)d_stride_bn, &stride_bn, sizeof(int));
    checkCudaErrors(err);
    err = hipMemcpyHtoD((hipDeviceptr_t)d_stride_cm, &stride_cm, sizeof(int));
    checkCudaErrors(err);
    err = hipMemcpyHtoD((hipDeviceptr_t)d_stride_cn, &stride_cn, sizeof(int));
    checkCudaErrors(err);

    // 设置内核参数
    void *args[] = { &d_A, &d_B, &d_C,&d_m,&d_n,&d_k,&d_stride_am,&d_stride_ak,&d_stride_bk,&d_stride_bn,&d_stride_cm,&d_stride_cn};
    
    // 启动内核
    err = hipModuleLaunchKernel(kernel,
                         1, 1, 1, // 网格维度
                         256, 1, 1, // 块维度
                         0, // 共享内存字节数
                         0, // 流
                         args, // 参数
                         0); // 额外参数
    if(err==hipSuccess) std::cout<<12<<' ';
    checkCudaErrors(err);

    // 将结果从设备复制回主机
    err = hipMemcpyDtoH(h_C, (hipDeviceptr_t)d_C, M * sizeof(float));
    if(err==hipSuccess) std::cout<<13<<' ';
    checkCudaErrors(err);

    // 打印结果
    for (int i = 0; i < 10; i++) { // 打印前10个结果
        std::cout << "C[" << i <<","<<i<<" ] = " << h_C[i] << std::endl;
    }

    // 释放设备内存
    err = hipFree((hipDeviceptr_t)d_A);
    if(err==hipSuccess) std::cout<<14<<' ';
    checkCudaErrors(err);
    err = hipFree((hipDeviceptr_t)d_B);
    checkCudaErrors(err);
    err = hipFree((hipDeviceptr_t)d_C);
    checkCudaErrors(err);
    err = hipFree((hipDeviceptr_t)d_m);
    checkCudaErrors(err);
    err = hipFree((hipDeviceptr_t)d_n);
    checkCudaErrors(err);
    err = hipFree((hipDeviceptr_t)d_k);
    checkCudaErrors(err);
    err = hipFree((hipDeviceptr_t)d_stride_am);
    checkCudaErrors(err);
    err = hipFree((hipDeviceptr_t)d_stride_ak);
    checkCudaErrors(err);
    err = hipFree((hipDeviceptr_t)d_stride_bk);
    checkCudaErrors(err);
    err = hipFree((hipDeviceptr_t)d_stride_bn);
    checkCudaErrors(err);
    err = hipFree((hipDeviceptr_t)d_stride_cm);
    checkCudaErrors(err);
    err = hipFree((hipDeviceptr_t)d_stride_cn);
    checkCudaErrors(err);

    err = hipCtxDestroy(context);
    if(err==hipSuccess) std::cout<<15<<' ';
    checkCudaErrors(err);
    return 0;
}