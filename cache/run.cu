#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <iostream>
#include <vector>

const int kCudaThreadsNum = 512;
inline int CudaGetBlocks(const int N)
{
	return (N + kCudaThreadsNum - 1) / kCudaThreadsNum;
}
#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n);  i += blockDim.x * gridDim.x)
__global__ void relu_gpu(float* in, float* out, int n)
{
	CUDA_KERNEL_LOOP(i, n) {
		out[i] = in[i] > 0 ? in[i] : 0;
	}
}
__global__ void relu_gpu_back(float* in, float* out, int n)
{
	CUDA_KERNEL_LOOP(i, n) {
		out[i] = in[i] > 0 ? 1 : 0;
	}
}
__global__ void sigmoid_gpu  (float* in, float* out, int n)
{
	CUDA_KERNEL_LOOP(i, n) {
		out[i] = 1 / (1 + exp(-in[i]));
	}
}
__global__ void sigmoid_gpu_back(float* in, float* out, int n)
{
	CUDA_KERNEL_LOOP(i, n) {
		out[i] = (1 / (1 + exp(-in[i]))) * (1 - 1 / (1 + exp(-in[i])));
	}
}
void print_array(std::vector<float> arr, int num, const char* name)
{
	printf("\n\n%s:\n", name);
	for (int i = 0; i < num; i++)
	{
		if (i % 8 == 0)
		{
			printf("\n");
		}
		printf("%.2f ", arr[i]);
	}
	printf("\n");
}
class Tensor
{
public:
	std::vector<float> shape;
	float* data;
	bool ongpu;
	std::vector<float> getshape()
	{
		return shape;
	}

	Tensor::Tensor(std::vector<float> vec, bool gpu)
	{
		int s = vec.size();
		shape = vec;
		ongpu = gpu;
		if (ongpu)
		{
			hipMalloc(&data, sizeof(float) * s);
		}
		else
		{
			data = new float[s];
		}
	}
	void freecuda()
	{
		if (ongpu) hipFree(data);
		else free(data);
	}

	Tensor cpu()
	{
		int s = this->shape.size();
		std::vector<float> cpu_tensor;
		cpu_tensor = shape;
		hipMemcpy(cpu_tensor.data(), this->data, s * sizeof(float), hipMemcpyDeviceToHost);
		Tensor temp(cpu_tensor, false);
		return temp;
	}
	Tensor gpu()
	{
		int s = this->shape.size();
		std::vector<float> gpu_tensor;
		gpu_tensor = shape;
		hipMemcpy(gpu_tensor.data(), this->data, s * sizeof(float), hipMemcpyHostToDevice);
		Tensor temp(gpu_tensor, true);
		return temp;
	}
};
int main()
{
	//在CPU上输入一个N个元素的数组
	const int N = 64;
	std::vector<float> h_in;
	h_in.reserve(N);
	for (int i = 0; i < N; i++)
	{
		h_in.push_back((i - 32) * 0.1);
	}
	Tensor hh_in(h_in, false);
	float* hin = (float*)malloc(N * sizeof(float));
	float* hout = (float*)malloc(N * sizeof(float));
	for (int i = 0; i < N; i++)
	{
		hin[i] = h_in[i];
	}
	print_array(h_in, N, "the data input on cpu");

	Tensor dd_in = hh_in.gpu();
	float* din;
	float* dout;
	hipMalloc(&din, N * sizeof(float));
	hipMalloc(&dout, N * sizeof(float));
	hipMemcpy(din, hin, N * sizeof(float), hipMemcpyHostToDevice);
	//relu_gpu << <1, N >> > (din, dout, N);
	//relu_gpu_back << <1, N >> > (din, dout, N);
	//sigmoid_gpu << <1, N >> > (din, dout, N);
	sigmoid_gpu_back << <1, N >> > (din, dout, N);

	std::vector<float> h_out;
	h_out.resize(N);
	hipMemcpy(h_out.data(), dout, N * sizeof(float), hipMemcpyDeviceToHost);
	print_array(h_out, N, "the data output on cpu");
	dd_in.freecuda();
	hh_in.freecuda();

	return 0;
}
