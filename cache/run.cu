#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define checkCudaErrors(err) { \
    if (err != hipSuccess) { \
        const char *errorName; \
        const char *errorString; \
        hipDrvGetErrorName(err, &errorName); \
        hipDrvGetErrorString(err, &errorString); \
        std::cerr << "CUDA Error: " << errorName << ": " << errorString << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

int main() {
    hipDevice_t device;
    hipCtx_t context;
    hipModule_t module;
    hipFunction_t kernel;
    hipError_t err;

    // Initialize the driver API
    err = hipInit(0);
    if(err!=hipSuccess) std::cout<<1;
    checkCudaErrors(err);

    // Get the first device
    err = hipDeviceGet(&device, 0);
    if(err!=hipSuccess) std::cout<<2;
    checkCudaErrors(err);

    // Create a context
    err = hipCtxCreate(&context, 0, device);
    if(err!=hipSuccess) std::cout<<3;
    checkCudaErrors(err);

    // Load the module
    err = hipModuleLoad(&module, "matmul_cache_kernel.cubin");
    if(err!=hipSuccess) std::cout<<4;
    checkCudaErrors(err);
     
    // Get the kernel function
    err = hipModuleGetFunction(&kernel, module, "matmul_cache_kernel_0d1d2d3d4d5d6d7c8d9c10d11c");
    if(err!=hipSuccess) std::cout<<5;
    checkCudaErrors(err);
CUfunctionAttributes attrs;
cuFuncGetAttributes(&attrs, kernel);


    return 0;
}