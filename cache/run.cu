#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define checkCudaErrors(err) { \
    if (err != hipSuccess) { \
        const char *errorName; \
        const char *errorString; \
        hipDrvGetErrorName(err, &errorName); \
        hipDrvGetErrorString(err, &errorString); \
        std::cerr << "CUDA Error: " << errorName << ": " << errorString << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

int main() {
    hipDevice_t device;
    hipCtx_t context;
    hipModule_t module;
    hipFunction_t kernel;
    hipError_t err;

    // Initialize the driver API
    err = hipInit(0);
    checkCudaErrors(err);

    // Get the first device
    err = hipDeviceGet(&device, 0);
    checkCudaErrors(err);

    // Create a context
    err = hipCtxCreate(&context, 0, device);
    checkCudaErrors(err);

    // Load the module
    err = hipModuleLoad(&module, "matmul_kernel.cubin");
    checkCudaErrors(err);

    // Get the kernel function
    err = hipModuleGetFunction(&kernel, module, "matmul_kernel");
    checkCudaErrors(err);

    // Launch the kernel
    void *args[] = { /* kernel arguments */ };
    err = hipModuleLaunchKernel(kernel, 1, 1, 1, 1, 1, 1, 0, 0, args, 0);
    checkCudaErrors(err);

    // Clean up
    hipModuleUnload(module);
    hipCtxDestroy(context);

    return 0;
}