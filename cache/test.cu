#include <iostream>
#include <hip/hip_runtime.h>


// 检查CUDA错误的宏
#define checkCudaErrors(err) { \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error: " << err << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

int main() {
    hipDevice_t device;
    hipCtx_t context;
    hipModule_t module;
    hipFunction_t kernel;
    hipError_t err;

    // 初始化CUDA驱动API
    err = hipInit(0);
    if(err==hipSuccess) std::cout<<1<<' ';
    checkCudaErrors(err);

    // 获取第一个设备
    err = hipDeviceGet(&device, 0);
    if(err==hipSuccess) std::cout<<2<<' ';
    checkCudaErrors(err);

    // 创建上下文
    err = hipCtxCreate(&context, 0, device);
    if(err==hipSuccess) std::cout<<3<<' ';
    checkCudaErrors(err);

    // 加载模块
    err = hipModuleLoad(&module, "matmul_cache_kernel.cubin");
    if(err==hipSuccess) std::cout<<4<<' ';
    checkCudaErrors(err);

    // 获取内核函数
    err = hipModuleGetFunction(&kernel, module, "matmul_cache_kernel_0d1d2d3d4d5d6d7c8d9c10d11c");
    if(err==hipSuccess) std::cout<<5<<' ';
    checkCudaErrors(err);

    // 定义输入和输出数据
    float h_A[512][512];
    float h_B[512][512];
    float h_C[512][512];
    float *d_A, *d_B, *d_C;

    int M=512;
    int N=512;
    int K=512;
    int *d_m,*d_n,*d_k;
   
    int stride_a[2];
    int stride_b[2];
    int stride_c[2];
    stride_a[0]=512;
    stride_a[1]=1;
    stride_b[0]=512;
    stride_b[1]=1;
    stride_c[0]=512;
    stride_c[1]=1;

    int *d_stride_a,*d_stride_b,*d_stride_c;

    // 初始化
    for (int i = 0; i < 512; i++) 
    {
        for(int j = 0;j < 512; j++)
            h_A[i][j] = static_cast<float>(i);       
    }
    for (int i = 0; i < 512; i++) 
    {
        for(int j = 0;j < 512; j++)
            h_B[i][j] = static_cast<float>(j);       
    }
    
    // 分配
    err = hipMalloc((hipDeviceptr_t*)&d_A, M*K * sizeof(float));
    if(err==hipSuccess) std::cout<<6<<' ';
    checkCudaErrors(err);
    err = hipMalloc((hipDeviceptr_t*)&d_B, K*N * sizeof(float));
    checkCudaErrors(err);
    err = hipMalloc((hipDeviceptr_t*)&d_C, M*N * sizeof(float));
    checkCudaErrors(err);

    err = hipMalloc((hipDeviceptr_t*)&d_m, sizeof(int));
    if(err==hipSuccess) std::cout<<7<<' ';
    checkCudaErrors(err);
    err = hipMalloc((hipDeviceptr_t*)&d_n, sizeof(int));
    checkCudaErrors(err);
    err = hipMalloc((hipDeviceptr_t*)&d_k, sizeof(int));
    checkCudaErrors(err);

    err = hipMalloc((hipDeviceptr_t*)&d_stride_a, 2*sizeof(int));
    if(err==hipSuccess) std::cout<<8<<' ';
    checkCudaErrors(err);
    err = hipMalloc((hipDeviceptr_t*)&d_stride_b, 2*sizeof(int));
    checkCudaErrors(err);
    err = hipMalloc((hipDeviceptr_t*)&d_stride_c, 2*sizeof(int));
    checkCudaErrors(err);


    // 复制
    err = hipMemcpyHtoD((hipDeviceptr_t)d_A, h_A, M*K * sizeof(float));
    if(err==hipSuccess) std::cout<<9<<' ';
    checkCudaErrors(err);
    err = hipMemcpyHtoD((hipDeviceptr_t)d_B, h_B, K*N * sizeof(float));
    checkCudaErrors(err);

    err = hipMemcpyHtoD((hipDeviceptr_t)d_m, &M, sizeof(int));
    if(err==hipSuccess) std::cout<<10<<' ';
    checkCudaErrors(err);
    err = hipMemcpyHtoD((hipDeviceptr_t)d_n, &N, sizeof(int));
    checkCudaErrors(err);
    err = hipMemcpyHtoD((hipDeviceptr_t)d_k, &K, sizeof(int));
    checkCudaErrors(err);

    err = hipMemcpyHtoD((hipDeviceptr_t)d_stride_a, &stride_a, sizeof(int));
    if(err==hipSuccess) std::cout<<11<<' ';
    checkCudaErrors(err);
    err = hipMemcpyHtoD((hipDeviceptr_t)d_stride_b, &stride_b, sizeof(int));
    checkCudaErrors(err);
    err = hipMemcpyHtoD((hipDeviceptr_t)d_stride_c, &stride_c, sizeof(int));
    checkCudaErrors(err);

    // 设置内核参数
    void *args[] = { d_A, d_B, d_C,d_m,d_n,d_k,d_stride_a,d_stride_b,d_stride_c};
    
    hipStream_t stream;
    hipStreamCreate(&stream);
    // 启动
    err = hipModuleLaunchKernel(kernel,
                         4, 1, 1, // 网格维度
                         32, 32, 32, // 块维度
                         65536, // 共享内存字节数
                         stream, // 流
                         args, // 参数
                         0); // 额外参数
    if(err==hipSuccess) std::cout<<12<<' ';
    checkCudaErrors(err);

    // 复制
    err = hipMemcpyDtoH(h_C, (hipDeviceptr_t)d_C, M*N * sizeof(float));
    if(err==hipSuccess) std::cout<<13<<' ';
    checkCudaErrors(err);

    // 打印
    for (int i = 0; i < 10; i++) { 
        std::cout << "C[" << i <<","<<i<<" ] = " << h_C[i][i] << std::endl;
    }

    // 释放
    err = hipFree((hipDeviceptr_t)d_A);
    if(err==hipSuccess) std::cout<<14<<' ';
    checkCudaErrors(err);
    err = hipFree((hipDeviceptr_t)d_B);
    checkCudaErrors(err);
    err = hipFree((hipDeviceptr_t)d_C);
    checkCudaErrors(err);
    err = hipFree((hipDeviceptr_t)d_m);
    checkCudaErrors(err);
    err = hipFree((hipDeviceptr_t)d_n);
    checkCudaErrors(err);
    err = hipFree((hipDeviceptr_t)d_k);
    checkCudaErrors(err);
    err = hipFree((hipDeviceptr_t)d_stride_a);
    checkCudaErrors(err);
    err = hipFree((hipDeviceptr_t)d_stride_b);
    checkCudaErrors(err);
    err = hipFree((hipDeviceptr_t)d_stride_c);
    checkCudaErrors(err);

    err = hipCtxDestroy(context);
    if(err==hipSuccess) std::cout<<15<<' ';
    checkCudaErrors(err);
    return 0;
}