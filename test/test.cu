#include <iostream>
#include <hip/hip_runtime.h>


// 检查CUDA错误的宏
#define checkCudaErrors(err) {\
    if(err != hipSuccess) {\
    const char *errorName;\
    const char *errorString;\
    hipDrvGetErrorName(err, &errorName);\
    hipDrvGetErrorString(err, &errorString);\
    std::cerr << "CUDA Error: " << errorName << ": " << errorString << std::endl;\
    exit(EXIT_FAILURE);}\
}

int main() {
    hipDevice_t device;
    hipCtx_t context;
    hipModule_t module;
    hipFunction_t kernel;
    hipError_t err;

    // 初始化CUDA驱动API
    err = hipInit(0);
    if(err==hipSuccess) std::cout<<1<<' ';
    checkCudaErrors(err);

    // 获取第一个设备
    err = hipDeviceGet(&device, 0);
    if(err==hipSuccess) std::cout<<2<<' ';
    checkCudaErrors(err);

    // 创建上下文
    err = hipCtxCreate(&context, 0, device);
    if(err==hipSuccess) std::cout<<3<<' ';
    checkCudaErrors(err);

    // 加载模块
    err = hipModuleLoad(&module, "matmul_cache_kernel.cubin");
    if(err==hipSuccess) std::cout<<4<<' ';
    checkCudaErrors(err);

    // 获取内核函数
    err = hipModuleGetFunction(&kernel, module, "matmul_cache_kernel_0d1d2d");
    if(err==hipSuccess) std::cout<<5<<' ';
    checkCudaErrors(err);

    // 定义输入和输出数据
    float h_A[512][512];
    float h_B[512][512];
    float h_C[512][512];
    float* d_A=nullptr;
    float* d_B=nullptr;
    float* d_C=nullptr;

    int M=512;
    int N=512;
    int K=512;
    // 初始化
    for (int i = 0; i < 512; i++) 
    {
        for(int j = 0;j < 512; j++)
            h_A[i][j] = 0.1;       
    }
    for (int i = 0; i < 512; i++) 
    {
        for(int j = 0;j < 512; j++)
            h_B[i][j] = 0.2;       
    }
    // 分配
    err = hipMalloc((hipDeviceptr_t*)&d_A, M*K * sizeof(float));
    if(err==hipSuccess) std::cout<<6<<' ';
    checkCudaErrors(err);
    err = hipMalloc((hipDeviceptr_t*)&d_B, K*N * sizeof(float));
    checkCudaErrors(err);
    err = hipMalloc((hipDeviceptr_t*)&d_C, M*N * sizeof(float));
    checkCudaErrors(err);
    // 复制
    err = hipMemcpyHtoD((hipDeviceptr_t)d_A, h_A, M*K * sizeof(float));
    if(err==hipSuccess) std::cout<<9<<' ';
    checkCudaErrors(err);
    err = hipMemcpyHtoD((hipDeviceptr_t)d_B, h_B, K*N * sizeof(float));
    checkCudaErrors(err);
    void *args[] = {(d_A), (d_B), (d_C)};
    void *d_args=nullptr;
    err = hipMalloc((hipDeviceptr_t*)&d_args, 3* sizeof(void*));
    err = hipMemcpyHtoD((hipDeviceptr_t)d_args, args,3* sizeof(void*));
    if(err==hipSuccess) std::cout<<9<<' ';
    checkCudaErrors(err);
    // 设置内核参数
    
    
    hipStream_t hstream;
    err = hipStreamCreateWithFlags(&hstream, hipStreamDefault);
    checkCudaErrors(err);
    if(err==hipSuccess) std::cout<<10<<' ';
    
    // 启动
    dim3 gridDim(1, 1, 1);
    dim3 blockDim(256, 1, 1);
    unsigned int  sharedMemBytes=49152;
    //kernel<<<gridDim, blockDim ,sharedMemBytes,hstream>>>(d_A,d_B,d_C);
    err = hipModuleLaunchKernel(kernel,
                         gridDim.x, gridDim.y, gridDim.z,
                         blockDim.x, blockDim.y, blockDim.z,
                         sharedMemBytes, // 共享内存字节数
                         hstream, // 流
                         &d_args, // 参数
                         NULL); // 额外参数
    if(err==hipSuccess) std::cout<<12<<' ';
    checkCudaErrors(err);

    // 复制
    err = hipMemcpyDtoH(h_C, (hipDeviceptr_t)d_C, M*N * sizeof(float));
    if(err==hipSuccess) std::cout<<13<<' ';
    checkCudaErrors(err);

    // 打印
    for (int i = 0; i < 10; i++) { 
        std::cout << "C[" << i <<","<<i<<" ] = " << h_C[i][i] << std::endl;
    }

    // 释放
    err = hipFree((hipDeviceptr_t)d_A);
    if(err==hipSuccess) std::cout<<14<<' ';
    checkCudaErrors(err);
    err = hipFree((hipDeviceptr_t)d_B);
    checkCudaErrors(err);
    err = hipFree((hipDeviceptr_t)d_C);
    checkCudaErrors(err);
    err = hipCtxDestroy(context);
    if(err==hipSuccess) std::cout<<15<<' ';
    checkCudaErrors(err);
    return 0;
}