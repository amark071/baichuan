#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, 0);

     if (err != hipSuccess) {
         // 打印错误信息
         std::cerr << "Error getting device properties: " << hipGetErrorString(err) << std::endl;
         return -1;
    }

     // 输出每个线程块的最大共享内存大小
    std::cout << "Maximum shared memory per block: " << prop.sharedMemPerBlock << " bytes" << std::endl;


    // 输出每个线程块的最大维度
    std::cout << "Maximum block dimensions:" << std::endl;
    std::cout << "  x: " << prop.maxThreadsDim[0] << std::endl;
    std::cout << "  y: " << prop.maxThreadsDim[1] << std::endl;
    std::cout << "  z: " << prop.maxThreadsDim[2] << std::endl;

    // 输出每个网格的最大维度
    std::cout << "Maximum grid dimensions:" << std::endl;
    std::cout << "  x: " << prop.maxGridSize[0] << std::endl;
    std::cout << "  y: " << prop.maxGridSize[1] << std::endl;
    std::cout << "  z: " << prop.maxGridSize[2] << std::endl;
    return 0;
}

