#include <iostream>
#include <hip/hip_runtime.h>
extern "C" {
    #include "kernel.da01dcef_0d1d2d34567891011.h"
}

int main() {
    hipDevice_t device;
    hipCtx_t context;
    hipError_t err;
    err = hipInit(0);
    err = hipDeviceGet(&device, 0);
    err = hipCtxCreate(&context, 0, device);
    float h_A[512][512];float h_B[512][512];float h_C[512][512];
    float* d_A=nullptr; float* d_B=nullptr; float* d_C=nullptr;
    int M=512;int N=512;int K=512;

    for (int i = 0; i < 512; i++){
        for(int j = 0;j < 512; j++)
            h_A[i][j] = 0.1;
  }
     for (int i = 0; i < 512; i++){
        for(int j = 0;j < 512; j++)
            h_B[i][j] = 0.2;
  }
    err = hipMalloc((hipDeviceptr_t*)&d_A, M*K * sizeof(float));
    err = hipMalloc((hipDeviceptr_t*)&d_B, M*K * sizeof(float));
    err = hipMalloc((hipDeviceptr_t*)&d_C, M*N * sizeof(float));
    err = hipMemcpyHtoD((hipDeviceptr_t)d_A, h_A, M*K * sizeof(float));
    err = hipMemcpyHtoD((hipDeviceptr_t)d_B, h_B, M*K * sizeof(float));
    hipStream_t hstream;
    err = hipStreamCreateWithFlags(&hstream, hipStreamDefault);
    kernel_da01dcef_0d1d2d34567891011(hstream, (hipDeviceptr_t)d_A, (hipDeviceptr_t)d_B, (hipDeviceptr_t)d_C,(int32_t) 512, (int32_t) 512, (int32_t) 512, (int32_t) 512, (int32_t) 1, (int32_t) 512, (int32_t) 1, (int32_t) 512, (int32_t) 1);
    err = hipMemcpyDtoH(h_C, (hipDeviceptr_t)d_C, M*N * sizeof(float));
    for (int i = 0; i < 10; i++) {
        std::cout << "C[" << i <<","<<i<<" ] = " << h_C[i][i] << std::endl;
    }
    err = hipFree((hipDeviceptr_t)d_A);
    err = hipFree((hipDeviceptr_t)d_B);
    err = hipFree((hipDeviceptr_t)d_C);
    err = hipCtxDestroy(context);
    return 0;
}
